#include "hip/hip_runtime.h"
#include "cuda_fgk.h"

using namespace CUDA_FGK;

HZIP_CUDA_HOST_DEVICE Node* FGKTree::createTree() {
	Node* root = (Node*)malloc(sizeof(Node));
	root->isRoot = true;
	root->isLeaf = true;
	root->isZero = true;

	root->left_child = NULL;
	root->right_child = NULL;
	root->parent = NULL;

	root->symbol = INVALID;
	root->value = 0;
	root->order = alphabet_size << 1;

	return root;
}

HZIP_CUDA_HOST_DEVICE Node* FGKTree::findReplaceNode(Node* currMax, Node* root) {
	Node* result = currMax;
	if (result->value > root->value && !root->isLeaf) {
		Node* greatestLeft = findReplaceNode(result, root->left_child);
		if (greatestLeft) result = greatestLeft;

		Node* greatestRight = findReplaceNode(result, root->right_child);
		if (greatestRight) result = greatestRight;
	}
	else if (result->value == root->value && root->order > result->order) {
		result = root;
	}

	return (result != currMax) ? result : NULL;
}

HZIP_CUDA_HOST_DEVICE void FGKTree::swapNodes(Node* x, Node* y) {
	HZIP_SIZE_T temp_order = x->order;
	x->order = y->order;
	y->order = temp_order;

	if (x->parent->left_child == x) {
		x->parent->left_child = y;
	}
	else if (x->parent->right_child == x) {
		x->parent->right_child = y;
	}

	if (y->parent->left_child == y) {
		y->parent->left_child = x;
	}
	else if (y->parent->right_child == y) {
		y->parent->right_child = x;
	}

	Node* temp_parent = x->parent;
	x->parent = y->parent;
	y->parent = temp_parent;

}

HZIP_CUDA_HOST_DEVICE Node* FGKTree::addChild(Node* parent, HZIP_SIZE_T symbol, HZIP_SIZE_T order, HZIP_SIZE_T value, bool isZero, bool isRoot) {
	Node* child = (Node*)malloc(sizeof(Node));
	child->isLeaf = true;
	child->isRoot = isRoot;
	child->isZero = isZero;
	child->order = order;
	child->symbol = symbol;
	child->parent = parent;
	child->value = value;
	return child;
}

HZIP_CUDA_HOST_DEVICE Node* FGKTree::addSymbol(HZIP_SIZE_T symbol) {
	Node* previousZeroNode = *zeroNode;
	Node* rightChild = addChild(*zeroNode, symbol, previousZeroNode->order - 1, 1, false, false);
	Node* leftChild = addChild(*zeroNode, INVALID, previousZeroNode->order - 2, 0, true, false);
	previousZeroNode->isLeaf = false;
	previousZeroNode->isZero = false;
	previousZeroNode->left_child = leftChild;
	previousZeroNode->right_child = rightChild;

	symbols[symbol] = (Symbol*)malloc(sizeof(Symbol));
	symbols[symbol]->symbol = symbol;
	symbols[symbol]->tree = rightChild;


	*zeroNode = leftChild;
	return previousZeroNode;
}

HZIP_CUDA_HOST_DEVICE void FGKTree::updateTree(Node *currNode) {
	while (!currNode->isRoot) {
		Node *replaceNode = findReplaceNode(currNode, *root);

		if (replaceNode && currNode->parent != replaceNode) {
			swapNodes(currNode, replaceNode);
		}

		(currNode->value)++;
		currNode = currNode->parent;
	}

	(currNode->value)++;
}

HZIP_CUDA_HOST_DEVICE void FGKTree::reverseCode(bool *code, HZIP_SIZE_T codeSize) {
	if (code == NULL) {
		return;
	}

	bool *start = code;
	bool *end = code + (codeSize - 1);

	while (start < end) {
		int temp = *start;
		*start = *end;
		*end = temp;
		start++;
		end--;
	}
}

HZIP_CUDA_HOST_DEVICE bool* FGKTree::codeOfNode(Node *node, HZIP_SIZE_T *n) {
	Node *current = node;
	 
	int i = 0;
	while (!current->isRoot) {
		Node *parent = current->parent;
		codebuffer[i] = (parent->left_child == current) ? 0 : 1;
		current = current->parent;
		i++;
	}
	reverseCode(codebuffer, i);

	*n = i;
	return codebuffer;
}

HZIP_CUDA_HOST_DEVICE Node* FGKTree::getTreeFromSymbol(HZIP_SIZE_T symbol) {
	Symbol *symbolPtr = symbols[symbol];

	if (!symbolPtr) {
		return NULL;
	}

	return symbolPtr->tree;
}

HZIP_CUDA_HOST_DEVICE FGKTree::FGKTree(HZIP_SIZE_T n) {
	alphabet_size = n;
	root = new Node*;
	zeroNode = new Node*;
	*root = createTree();
	*zeroNode = *root;
	symbols = (Symbol**)malloc(sizeof(Symbol*) * n);
	memset(symbols, 0, n);
	codebuffer = (bool*)malloc(sizeof(int) * alphabet_size * 2);
}

void FGKTree::encode(HZIP_SIZE_T symbol, bool** code, HZIP_SIZE_T *code_length) {
	Node* node = getTreeFromSymbol(symbol);
	if (node) {
		*code = codeOfNode(node, code_length);
		updateTree(node);
	}
	else {
		*code = codeOfNode(*zeroNode, code_length);
		Node* newNode = addSymbol(symbol);
		updateTree(newNode);

	}

}


/*
__global__ void fgk_kernel() {
	CUDA_FGK::FGKTree tree(0x100);
	HZIP_SIZE_T *n = (HZIP_SIZE_T*)malloc(sizeof(HZIP_SIZE_T));
	bool **code = (bool**)malloc(sizeof(bool*));
	int i;
	for (i = 0; i < 1024; i++) {
		tree.encode(i % 2, code, n);
	}
}

void launch_fgk_benchmark_kernel(int n) {
	std::cout << "Running FGK CUDA benchmark ..." << std::endl;
	auto start = std::chrono::high_resolution_clock::now();
	fgk_kernel << <1, n >> > ();
	hipDeviceSynchronize();
	std::cout << "Time taken: " << (float)(std::chrono::high_resolution_clock::now() - start).count() / 1000000000 << " s" << std::endl;
}
*/